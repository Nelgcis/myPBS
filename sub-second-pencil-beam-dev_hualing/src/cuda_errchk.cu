/**
 * \file
 * \brief CUDA error assert function implementations
 */
#include "cuda_errchk.cuh"
#include "hip/hip_runtime_api.h"

#include <sstream>
#include <sstream>
#include <stdexcept>

void cudaAssert(const hipError_t code, const char* file, const int line, const bool abort)
{
    if (code != hipSuccess)
    {
        //fprintf(stderr,"cudaAssert: %s %s %d\n", hipGetErrorString(code), file, line);
        //if (abort) exit(code);
        std::ostringstream msgStream;
        msgStream << "cudaAssert: " << hipGetErrorString(code) << " " << file << ", line: " <<  line;
        throw std::runtime_error(msgStream.str());
    }
}
