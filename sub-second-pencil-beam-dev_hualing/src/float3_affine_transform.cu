#include "hip/hip_runtime.h"
/**
 * \file
 * \brief Float3AffineTransform class implementation
 */
#include "float3_affine_transform.cuh"
#include "hip/hip_vector_types.h"
#include "helper_float3.cuh"
#include <iostream>

Float3AffineTransform::Float3AffineTransform() : m(1.0f, 1.0f, 1.0f), v(make_float3(0.0f, 0.0f, 0.0f)) {}

Float3AffineTransform::Float3AffineTransform(const Matrix3x3 mIn, const float3 vIn) : m(mIn), v(vIn) {}

Float3AffineTransform::Float3AffineTransform(const Float3AffineTransform& in) : Float3AffineTransform(in.getMatrix(),in.getOffset()) {}

CUDA_CALLABLE_MEMBER float3 Float3AffineTransform::transformPoint(const float3 point) const {return m*point + v;}

CUDA_CALLABLE_MEMBER float3 Float3AffineTransform::transformVector(const float3 vector) const {return m*vector;}

Float3AffineTransform Float3AffineTransform::inverse() const
{
    return Float3AffineTransform(m.inverse(), m.inverse()*(v*(-1)));
}

void Float3AffineTransform::oneBasedToZeroBased(const bool toIdx)
{
    if (toIdx) v -= make_float3(1.0f, 1.0f, 1.0f);
    else v += make_float3(sum_float3(m.row0()), sum_float3(m.row1()), sum_float3(m.row2()));
}

CUDA_CALLABLE_MEMBER Matrix3x3 Float3AffineTransform::getMatrix() const {return m;}

CUDA_CALLABLE_MEMBER float3 Float3AffineTransform::getOffset() const {return v;}

void Float3AffineTransform::print() const
{
    printf("%f %f %f    %f\n", m.row0().x, m.row0().y, m.row0().z, v.x);
    printf("%f %f %f    %f\n", m.row1().x, m.row1().y, m.row1().z, v.y);
    printf("%f %f %f    %f\n", m.row2().x, m.row2().y, m.row2().z, v.z);
}

Float3AffineTransform concatFloat3AffineTransform(const Float3AffineTransform t1, const Float3AffineTransform t2)
{
    return(Float3AffineTransform(t2.m*t1.m, t2.m*t1.v+t2.v));
}

